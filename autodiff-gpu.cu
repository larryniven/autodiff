#include "hip/hip_runtime.h"
#include "autodiff/autodiff-gpu.h"
#include "autodiff/autodiff-op-gpu.h"
#include "la/la-gpu.h"

namespace autodiff {

    namespace gpu {

        void mul_eval(std::shared_ptr<op_t> t)
        {
            auto& A = get_output<la::gpu::matrix<double>>(get_child(t, 0));
            auto& v = get_output<la::gpu::vector<double>>(get_child(t, 1));

            if (t->output == nullptr) {
                la::gpu::vector<double> u;
                u.resize(A.rows());
                t->output = std::make_shared<la::gpu::vector<double>>(u);
            } else {
                auto& u = get_output<la::gpu::vector<double>>(t);
                la::gpu::zero(u);
            }

            auto& u = get_output<la::gpu::vector<double>>(t);
            la::gpu::mul(u, A, v);
        }

        void mul_grad(std::shared_ptr<op_t> t)
        {
            auto& grad = get_grad<la::gpu::vector<double>>(t);

            auto A_o = get_child(t, 0);
            auto v_o = get_child(t, 1);

            auto& A = get_output<la::gpu::matrix<double>>(A_o);
            auto& v = get_output<la::gpu::vector<double>>(v_o);

            if (A_o->grad == nullptr) {
                A_o->grad = std::make_shared<la::gpu::matrix<double>>(la::gpu::matrix<double>());
            }

            if (v_o->grad == nullptr) {
                v_o->grad = std::make_shared<la::gpu::vector<double>>(la::gpu::vector<double>());
            }

            auto& A_grad = get_grad<la::gpu::matrix<double>>(A_o);
            auto& v_grad = get_grad<la::gpu::vector<double>>(v_o);

            autodiff::op::gpu::iouter_prod(A_grad, grad, v);
            autodiff::op::gpu::ilmul(v_grad, A, grad);
        }

        void emul_eval(std::shared_ptr<op_t> t)
        {
            auto& u = get_output<la::gpu::vector<double>>(get_child(t, 0));
            auto& v = get_output<la::gpu::vector<double>>(get_child(t, 1));

            if (t->output == nullptr) {
                la::gpu::vector<double> z;
                z.resize(u.size());
                t->output = std::make_shared<la::gpu::vector<double>>(z);
            } else {
                auto& z = get_output<la::gpu::vector<double>>(t);
                la::gpu::zero(z);
            }

            auto& z = get_output<la::gpu::vector<double>>(t);
            la::gpu::emul(z, u, v);
        }

        void emul_grad(std::shared_ptr<op_t> t)
        {
            auto& grad = get_grad<la::gpu::vector<double>>(t);

            auto u_o = get_child(t, 0);
            auto v_o = get_child(t, 1);

            auto& u = get_output<la::gpu::vector<double>>(u_o);
            auto& v = get_output<la::gpu::vector<double>>(v_o);

            if (u_o->grad == nullptr) {
                u_o->grad = std::make_shared<la::gpu::vector<double>>(la::gpu::vector<double>());
            }

            if (v_o->grad == nullptr) {
                v_o->grad = std::make_shared<la::gpu::vector<double>>(la::gpu::vector<double>());
            }

            auto& u_grad = get_grad<la::gpu::vector<double>>(u_o);
            u_grad.resize(u.size());
            auto& v_grad = get_grad<la::gpu::vector<double>>(v_o);
            v_grad.resize(v.size());

            la::gpu::emul(u_grad, grad, v);
            la::gpu::emul(v_grad, grad, u);
        }

        void logistic_eval(std::shared_ptr<op_t> t)
        {
            auto& v = get_output<la::gpu::vector<double>>(get_child(t, 0));

            if (t->output == nullptr) {
                la::gpu::vector<double> z;
                z.resize(v.size());
                t->output = std::make_shared<la::gpu::vector<double>>(z);
            } else {
                auto& z = get_output<la::gpu::vector<double>>(t);
                la::gpu::zero(z);
            }

            auto& z = get_output<la::gpu::vector<double>>(t);
            autodiff::op::gpu::logistic(z, v);
        }

        void logistic_grad(std::shared_ptr<op_t> t)
        {
            auto& grad = get_grad<la::gpu::vector<double>>(t);
            auto& output = get_output<la::gpu::vector<double>>(t);

            auto ch = get_child(t, 0);
            if (ch->grad == nullptr) {
                ch->grad = std::make_shared<la::gpu::vector<double>>(la::gpu::vector<double>());
            }

            auto& result = get_grad<la::gpu::vector<double>>(ch);
            autodiff::op::gpu::ilogistic_grad(result, grad, output);
        }

        void relu_eval(std::shared_ptr<op_t> t)
        {
            auto& v = get_output<la::gpu::vector<double>>(get_child(t, 0));

            if (t->output == nullptr) {
                la::gpu::vector<double> z;
                z.resize(v.size());
                t->output = std::make_shared<la::gpu::vector<double>>(z);
            } else {
                auto& z = get_output<la::gpu::vector<double>>(t);
                la::gpu::zero(z);
            }

            auto& z = get_output<la::gpu::vector<double>>(t);
            autodiff::op::gpu::relu(z, v);
        }

        void relu_grad(std::shared_ptr<op_t> t)
        {
            auto& output = get_output<la::gpu::vector<double>>(t);
            auto& grad = get_grad<la::gpu::vector<double>>(t);

            auto ch = get_child(t, 0);
            if (ch->grad == nullptr) {
                ch->grad = std::make_shared<la::gpu::vector<double>>(la::gpu::vector<double>());
            }

            auto& result = get_grad<la::gpu::vector<double>>(ch);
            autodiff::op::gpu::irelu_grad(result, grad, output);
        }

        void tanh_eval(std::shared_ptr<op_t> t)
        {
            auto& v = get_output<la::gpu::vector<double>>(get_child(t, 0));

            if (t->output == nullptr) {
                la::gpu::vector<double> z;
                z.resize(v.size());
                t->output = std::make_shared<la::gpu::vector<double>>(z);
            } else {
                auto& z = get_output<la::gpu::vector<double>>(t);
                la::gpu::zero(z);
            }

            auto& z = get_output<la::gpu::vector<double>>(t);
            autodiff::op::gpu::tanh(z, v);
        }

        void tanh_grad(std::shared_ptr<op_t> t)
        {
            auto& grad = get_grad<la::gpu::vector<double>>(t);
            auto& output = get_output<la::gpu::vector<double>>(t);

            auto ch = get_child(t, 0);
            if (ch->grad == nullptr) {
                ch->grad = std::make_shared<la::gpu::vector<double>>(la::gpu::vector<double>());
            }

            auto& result = get_grad<la::gpu::vector<double>>(ch);
            autodiff::op::gpu::itanh_grad(result, grad, output);
        }

        void add_eval(std::shared_ptr<op_t> t)
        {
            auto& g = *t->graph;

            assert(g.adj[t->id].size() > 0);

#ifndef NDEBUG
            for (int i = 1; i < g.adj[t->id].size(); ++i) {
                assert(get_output<la::gpu::vector<double>>(get_child(t, i-1)).size()
                    == get_output<la::gpu::vector<double>>(get_child(t, i)).size());
            }
#endif

            if (t->output == nullptr) {
                la::gpu::vector<double> z;
                z.resize(get_output<la::gpu::vector<double>>(get_child(t, 0)).size());
                t->output = std::make_shared<la::gpu::vector<double>>(z);
            } else {
                auto& z = get_output<la::gpu::vector<double>>(t);
                la::gpu::zero(z);
            }

            auto& result = get_output<la::gpu::vector<double>>(t);

            for (int i = 0; i < g.adj[t->id].size(); ++i) {
                auto& u = get_output<la::gpu::vector<double>>(get_child(t, i));

                la::gpu::iadd(result, u);
            }
        }

        void add_grad(std::shared_ptr<op_t> t)
        {
            auto& g = *t->graph;

            auto& grad = get_grad<la::gpu::vector<double>>(t);

            for (int i = 0; i < g.adj[t->id].size(); ++i) {
                auto c = get_child(t, i);

                if (c->grad == nullptr) {
                    c->grad = std::make_shared<la::gpu::vector<double>>(la::gpu::vector<double>());
                }

                auto& u = get_grad<la::gpu::vector<double>>(c);
                u.resize(grad.size());

                la::gpu::iadd(u, grad);
            }
        }

        void softmax_eval(std::shared_ptr<op_t> t)
        {
            auto& v = get_output<la::gpu::vector<double>>(get_child(t, 0));

            if (t->output == nullptr) {
                la::gpu::vector<double> z;
                z.resize(v.size());
                t->output = std::make_shared<la::gpu::vector<double>>(z);
            } else {
                auto& z = get_output<la::gpu::vector<double>>(t);
                la::gpu::zero(z);
            }

            auto& z = get_output<la::gpu::vector<double>>(t);
            autodiff::op::gpu::softmax(z, v);
        }

        void softmax_grad(std::shared_ptr<op_t> t)
        {
            auto& output = get_output<la::gpu::vector<double>>(t);
            auto& grad = get_grad<la::gpu::vector<double>>(t);

            auto ch = get_child(t, 0);
            if (ch->grad == nullptr) {
                ch->grad = std::make_shared<la::gpu::vector<double>>(la::gpu::vector<double>());
            }

            auto& result = get_grad<la::gpu::vector<double>>(ch);
            autodiff::op::gpu::isoftmax_grad(result, grad, output);
        }

        void logsoftmax_eval(std::shared_ptr<op_t> t)
        {
            auto& v = get_output<la::gpu::vector<double>>(get_child(t, 0));

            if (t->output == nullptr) {
                la::gpu::vector<double> z;
                z.resize(v.size());
                t->output = std::make_shared<la::gpu::vector<double>>(z);
            } else {
                auto& z = get_output<la::gpu::vector<double>>(t);
                la::gpu::zero(z);
            }

            auto& z = get_output<la::gpu::vector<double>>(t);
            autodiff::op::gpu::logsoftmax(z, v);
        }

        void logsoftmax_grad(std::shared_ptr<op_t> t)
        {
            auto& output = get_output<la::gpu::vector<double>>(t);
            auto& grad = get_grad<la::gpu::vector<double>>(t);

            auto ch = get_child(t, 0);
            if (ch->grad == nullptr) {
                ch->grad = std::make_shared<la::gpu::vector<double>>(la::gpu::vector<double>());
            }

            auto& result = get_grad<la::gpu::vector<double>>(ch);
            autodiff::op::gpu::ilogsoftmax_grad(result, grad, output);
        }

        void dot_eval(std::shared_ptr<op_t> t)
        {
            auto& v = get_output<la::gpu::vector<double>>(get_child(t, 0));
            auto& u = get_output<la::gpu::vector<double>>(get_child(t, 1));

            t->output = std::make_shared<double>(la::gpu::dot(v, u));
        }

        void dot_grad(std::shared_ptr<op_t> t)
        {
            auto c0 = get_child(t, 0);
            auto c1 = get_child(t, 1);

            auto& v = get_output<la::gpu::vector<double>>(c0);
            auto& u = get_output<la::gpu::vector<double>>(c1);

            assert(v.size() == u.size());

            double grad = get_grad<double>(t);

            if (c0->grad == nullptr) {
                c0->grad = std::make_shared<la::gpu::vector<double>>(la::gpu::vector<double>());
            }

            auto& v_grad = get_grad<la::gpu::vector<double>>(c0);
            v_grad.resize(u.size());

            hipblasDaxpy(la::gpu::device::get_handle(), v_grad.size(), &grad, u.data(), 1, v_grad.data(), 1);

            if (c1->grad == nullptr) {
                c1->grad = std::make_shared<la::gpu::vector<double>>(la::gpu::vector<double>());
            }

            auto& u_grad = get_grad<la::gpu::vector<double>>(c1);
            u_grad.resize(v.size());

            hipblasDaxpy(la::gpu::device::get_handle(), u_grad.size(), &grad, v.data(), 1, u_grad.data(), 1);
        }

    }
}
