#include "hip/hip_runtime.h"
#include "autodiff/autodiff-gpu.h"
#include "autodiff/autodiff-op-gpu.h"
#include "la/la-gpu.h"
#include <thrust/execution_policy.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <thrust/for_each.h>
#include <algorithm>

namespace autodiff {

    namespace gpu {

        void weak_var_eval(std::shared_ptr<op_t> t)
        {
            int shift;
            std::vector<unsigned int> sizes;

            std::tie(shift, sizes) = *std::static_pointer_cast<
                std::pair<int, std::vector<unsigned int>>>(t->data);

            auto ch = get_child(t, 0);

            auto& v = get_output<la::gpu::tensor_like<double>>(ch);
            la::gpu::weak_tensor<double> w_v { v.data() + shift, sizes };
            t->output = std::make_shared<la::gpu::weak_tensor<double>>(w_v);

            if (ch->grad_needed && ch->grad == nullptr) {
                la::gpu::tensor<double> g;
                g.resize(v.sizes());
                ch->grad = std::make_shared<la::gpu::tensor<double>>(std::move(g));
            }

            if (ch->grad_needed) {
                auto& z = get_grad<la::gpu::tensor_like<double>>(ch);
                la::gpu::weak_tensor<double> w_z { z.data() + shift, sizes };
                t->grad = std::make_shared<la::gpu::weak_tensor<double>>(w_z);
            }
        }

        void weak_var_grad(std::shared_ptr<op_t> t)
        {
        }

        struct subtensor_op {

            double *c_data;
            double const *a_data;
            unsigned int dim;
            unsigned int const *shift;
            unsigned int const *sizes;
            unsigned int const *a_sizes;

            __device__
            void index_to_coord(int index, double *coord)
            {
                int i = index;

                for (int d = dim - 1; d > 0; --d) {
                    int c = i % sizes[d];
                    coord[d] = c;
                    i = (i - c) / sizes[d];
                }

                coord[0] = i;
            }

            __device__
            int coord_to_index(double *coord)
            {
                int result = 0;

                for (int d = 0; d < dim; ++d) {
                    result *= a_sizes[d];
                    result += coord[d];
                }

                return result;
            }

            __device__
            void operator()(int i)
            {
                double *coord = (double*) malloc(dim);

                for (int i = 0; i < dim; ++i) {
                    coord[i] = 0;
                }

                index_to_coord(i, coord);

                for (int j = 0; j < dim; ++j) {
                    coord[j] += shift[j];
                }

                int j = coord_to_index(coord);

                printf("%d %d\n", i, j);

                c_data[i] += a_data[j];

                free(coord);
            }

        };

        void subtensor_eval(std::shared_ptr<op_t> t)
        {
            auto& a = get_output<la::gpu::tensor_like<double>>(get_child(t, 0));

            std::vector<unsigned int> shift;
            std::vector<unsigned int> sizes;

            std::tie(shift, sizes) = *std::static_pointer_cast<std::pair<std::vector<unsigned int>,
                std::vector<unsigned int>>>(t->data);

            assert(a.dim() == shift.size());

            for (int i = 0; i < shift.size(); ++i) {
                assert(shift[i] + sizes[i] <= a.size(i));
            }

            if (t->output == nullptr) {
                la::gpu::tensor<double> c;
                c.resize(sizes);
                t->output = std::make_shared<la::gpu::tensor<double>>(std::move(c));
            }

            auto& c = get_output<la::gpu::tensor_like<double>>(t);

            thrust::counting_iterator<int> c_begin {0};
            thrust::counting_iterator<int> c_end = c_begin + c.vec_size();

            la::gpu::vector<unsigned int> dshift { la::cpu::vector<unsigned int>(shift) };
            la::gpu::vector<unsigned int> dsizes { la::cpu::vector<unsigned int>(sizes) };
            la::gpu::vector<unsigned int> dasizes { la::cpu::vector<unsigned int>(a.sizes()) };

            thrust::for_each(c_begin, c_end,
                subtensor_op { c.data(), a.data(), dshift.size(),
                    dshift.data(), dsizes.data(), dasizes.data() });
        }

        void subtensor_grad(std::shared_ptr<op_t> t)
        {
            auto ch = get_child(t, 0);

            auto& a = get_output<la::gpu::tensor_like<double>>(ch);

            std::vector<unsigned int> shift;
            std::vector<unsigned int> sizes;

            std::tie(shift, sizes) = *std::static_pointer_cast<std::pair<std::vector<unsigned int>,
                std::vector<unsigned int>>>(t->data);

            if (ch->grad == nullptr) {
                la::gpu::tensor<double> c;
                c.resize(a.sizes());
                ch->grad = std::make_shared<la::gpu::tensor<double>>(std::move(c));
            }

            auto& ch_grad = get_grad<la::gpu::tensor_like<double>>(ch);
            auto& t_grad = get_grad<la::gpu::tensor_like<double>>(t);

            thrust::counting_iterator<int> c_begin {0};
            thrust::counting_iterator<int> c_end = c_begin + t_grad.vec_size();

            la::gpu::vector<unsigned int> dshift { la::cpu::vector<unsigned int>(shift) };
            la::gpu::vector<unsigned int> dsizes { la::cpu::vector<unsigned int>(sizes) };
            la::gpu::vector<unsigned int> dasizes { la::cpu::vector<unsigned int>(a.sizes()) };

            thrust::for_each(c_begin, c_end,
                subtensor_op { ch_grad.data(), t_grad.data(), dshift.size(),
                    dshift.data(), dsizes.data(), dasizes.data() });
        }

        void mul_eval(std::shared_ptr<op_t> t)
        {
            auto& a = get_output<la::gpu::tensor_like<double>>(get_child(t, 0));
            auto& b = get_output<la::gpu::tensor_like<double>>(get_child(t, 1));

            if (t->output == nullptr) {
                la::gpu::tensor<double> c;

                std::vector<unsigned int> sizes = a.sizes();
                sizes.pop_back();
                sizes.push_back(b.size(b.dim() - 1));

                c.resize(sizes);

                t->output = std::make_shared<la::gpu::tensor<double>>(std::move(c));
            }

            auto& c = get_output<la::gpu::tensor_like<double>>(t);

            la::gpu::mul(c, a, b);
        }

        void mul_grad(std::shared_ptr<op_t> t)
        {
            auto& grad = get_grad<la::gpu::tensor_like<double>>(t);

            auto a_o = get_child(t, 0);
            auto b_o = get_child(t, 1);

            auto& a = get_output<la::gpu::tensor_like<double>>(a_o);
            auto& b = get_output<la::gpu::tensor_like<double>>(b_o);

            if (a_o->grad_needed && a_o->grad == nullptr) {
                la::gpu::tensor<double> g;
                la::gpu::resize_as(g, a);
                a_o->grad = std::make_shared<la::gpu::tensor<double>>(std::move(g));
            }

            if (b_o->grad_needed && b_o->grad == nullptr) {
                la::gpu::tensor<double> g;
                la::gpu::resize_as(g, b);
                b_o->grad = std::make_shared<la::gpu::tensor<double>>(std::move(g));
            }

            auto& a_grad = get_grad<la::gpu::tensor_like<double>>(a_o);
            auto& b_grad = get_grad<la::gpu::tensor_like<double>>(b_o);

            if (a_o->grad_needed) {
                la::gpu::rtmul(a_grad, grad, b);
            }

            if (b_o->grad_needed) {
                la::gpu::ltmul(b_grad, a, grad);
            }
        }

        void emul_eval(std::shared_ptr<op_t> t)
        {
            auto& u = get_output<la::gpu::tensor_like<double>>(get_child(t, 0));
            auto& v = get_output<la::gpu::tensor_like<double>>(get_child(t, 1));

            if (t->output == nullptr) {
                la::gpu::tensor<double> z;
                la::gpu::resize_as(z, u);
                t->output = std::make_shared<la::gpu::tensor<double>>(std::move(z));
            }

            auto& z = get_output<la::gpu::tensor_like<double>>(t);
            la::gpu::emul(z, u, v);
        }

        void emul_grad(std::shared_ptr<op_t> t)
        {
            auto& grad = get_grad<la::gpu::tensor_like<double>>(t);

            auto u_o = get_child(t, 0);
            auto v_o = get_child(t, 1);

            auto& u = get_output<la::gpu::tensor_like<double>>(u_o);
            auto& v = get_output<la::gpu::tensor_like<double>>(v_o);

            if (u_o->grad_needed && u_o->grad == nullptr) {
                la::gpu::tensor<double> g;
                la::gpu::resize_as(g, u);
                u_o->grad = std::make_shared<la::gpu::tensor<double>>(std::move(g));
            }

            if (v_o->grad_needed && v_o->grad == nullptr) {
                la::gpu::tensor<double> g;
                la::gpu::resize_as(g, v);
                v_o->grad = std::make_shared<la::gpu::tensor<double>>(std::move(g));
            }

            auto& u_grad = get_grad<la::gpu::tensor_like<double>>(u_o);
            auto& v_grad = get_grad<la::gpu::tensor_like<double>>(v_o);

            if (u_o->grad_needed) {
                la::gpu::emul(u_grad, grad, v);
            }

            if (v_o->grad_needed) {
                la::gpu::emul(v_grad, grad, u);
            }
        }

        void logistic_eval(std::shared_ptr<op_t> t)
        {
            auto& v = get_output<la::gpu::tensor_like<double>>(get_child(t, 0));

            if (t->output == nullptr) {
                la::gpu::tensor<double> z;
                la::gpu::resize_as(z, v);
                t->output = std::make_shared<la::gpu::tensor<double>>(std::move(z));
            }

            auto& z = get_output<la::gpu::tensor_like<double>>(t);
            op::gpu::logistic(z, v);
        }

        void logistic_grad(std::shared_ptr<op_t> t)
        {
            auto& grad = get_grad<la::gpu::tensor_like<double>>(t);
            auto& output = get_output<la::gpu::tensor_like<double>>(t);

            auto ch = get_child(t, 0);
            auto& ch_t = get_output<la::gpu::tensor_like<double>>(ch);

            if (ch->grad_needed && ch->grad == nullptr) {
                la::gpu::tensor<double> g;
                la::gpu::resize_as(g, ch_t);
                ch->grad = std::make_shared<la::gpu::tensor<double>>(std::move(g));
            }

            auto& result = get_grad<la::gpu::tensor_like<double>>(ch);

            if (ch->grad_needed) {
                op::gpu::ilogistic_grad(result, grad, output);
            }
        }

        void tanh_eval(std::shared_ptr<op_t> t)
        {
            auto& v = get_output<la::gpu::tensor_like<double>>(get_child(t, 0));

            if (t->output == nullptr) {
                la::gpu::tensor<double> z;
                la::gpu::resize_as(z, v);
                t->output = std::make_shared<la::gpu::tensor<double>>(std::move(z));
            }

            auto& z = get_output<la::gpu::tensor_like<double>>(t);
            op::gpu::tanh(z, v);
        }

        void tanh_grad(std::shared_ptr<op_t> t)
        {
            auto& grad = get_grad<la::gpu::tensor_like<double>>(t);
            auto& output = get_output<la::gpu::tensor_like<double>>(t);

            auto ch = get_child(t, 0);
            auto& ch_t = get_output<la::gpu::tensor_like<double>>(ch);

            if (ch->grad_needed && ch->grad == nullptr) {
                la::gpu::tensor<double> g;
                la::gpu::resize_as(g, ch_t);
                ch->grad = std::make_shared<la::gpu::tensor<double>>(std::move(g));
            }

            auto& result = get_grad<la::gpu::tensor_like<double>>(ch);

            if (ch->grad_needed) {
                op::gpu::itanh_grad(result, grad, output);
            }
        }

        void add_eval(std::shared_ptr<op_t> t)
        {
            auto& g = *t->graph;

            assert(g.adj[t->id].size() > 0);

            for (int i = 1; i < g.adj[t->id].size(); ++i) {
                if (get_output<la::gpu::tensor_like<double>>(get_child(t, i-1)).vec_size()
                        != get_output<la::gpu::tensor_like<double>>(get_child(t, i)).vec_size())
                {
                    std::cerr << get_output<la::gpu::tensor_like<double>>(get_child(t, i-1)).vec_size()
                        << " != " << get_output<la::gpu::tensor_like<double>>(
                            get_child(t, i)).vec_size() << std::endl;
                    exit(1);
                }
            }

            if (t->output == nullptr) {
                la::gpu::tensor<double> z;
                la::gpu::tensor_like<double>& m = get_output<la::gpu::tensor_like<double>>(
                    get_child(t, 0));
                la::gpu::resize_as(z, m);
                t->output = std::make_shared<la::gpu::tensor<double>>(std::move(z));
            }

            auto& result = get_output<la::gpu::tensor_like<double>>(t);

            for (int i = 0; i < g.adj[t->id].size(); ++i) {
                auto& u = get_output<la::gpu::tensor_like<double>>(get_child(t, i));

                la::gpu::axpy(result, 1, u);
            }
        }

        void add_grad(std::shared_ptr<op_t> t)
        {
            auto& g = *t->graph;

            auto& grad = get_grad<la::gpu::tensor_like<double>>(t);

            for (int i = 0; i < g.adj[t->id].size(); ++i) {
                auto c = get_child(t, i);

                if (c->grad_needed && c->grad == nullptr) {
                    auto& c_t = get_output<la::gpu::tensor_like<double>>(c);
                    la::gpu::tensor<double> g;
                    la::gpu::resize_as(g, c_t);
                    c->grad = std::make_shared<la::gpu::tensor<double>>(std::move(g));
                }

                auto& u = get_grad<la::gpu::tensor_like<double>>(c);

                if (c->grad_needed) {
                    la::gpu::axpy(u, 1, grad);
                }
            }
        }

        void sub_eval(std::shared_ptr<op_t> t)
        {
            auto& u = get_output<la::gpu::tensor_like<double>>(get_child(t, 0));
            auto& v = get_output<la::gpu::tensor_like<double>>(get_child(t, 1));

            if (t->output == nullptr) {
                la::gpu::tensor<double> z;
                la::gpu::resize_as(z, u);
                t->output = std::make_shared<la::gpu::tensor<double>>(std::move(z));
            }

            auto& result = get_output<la::gpu::tensor_like<double>>(t);
            la::gpu::copy(result, u);
            la::gpu::axpy(result, -1, v);
        }

        void sub_grad(std::shared_ptr<op_t> t)
        {
            auto& grad = get_grad<la::gpu::tensor_like<double>>(t);

            auto u_o = get_child(t, 0);
            auto& u_t = get_output<la::gpu::tensor_like<double>>(u_o);

            if (u_o->grad_needed && u_o->grad == nullptr) {
                la::gpu::tensor<double> g;
                la::gpu::resize_as(g, u_t);
                u_o->grad = std::make_shared<la::gpu::tensor<double>>(std::move(g));
            }

            auto v_o = get_child(t, 1);
            auto& v_t = get_output<la::gpu::tensor_like<double>>(v_o);

            if (v_o->grad_needed && v_o->grad == nullptr) {
                la::gpu::tensor<double> g;
                la::gpu::resize_as(g, v_t);
                v_o->grad = std::make_shared<la::gpu::tensor<double>>(std::move(g));
            }

            auto& u_grad = get_grad<la::gpu::tensor_like<double>>(u_o);
            auto& v_grad = get_grad<la::gpu::tensor_like<double>>(v_o);

            if (u_o->grad_needed) {
                la::gpu::axpy(u_grad, 1, grad);
            }

            if (v_o->grad_needed) {
                la::gpu::axpy(v_grad, -1, grad);
            }
        }

        void logsoftmax_eval(std::shared_ptr<op_t> t)
        {
            auto& v = get_output<la::gpu::tensor_like<double>>(get_child(t, 0));

            if (t->output == nullptr) {
                la::gpu::tensor<double> z;
                la::gpu::resize_as(z, v);
                t->output = std::make_shared<la::gpu::tensor<double>>(std::move(z));
            }

            auto& z = get_output<la::gpu::tensor_like<double>>(t);
            op::gpu::logsoftmax(z, v);
        }

        void logsoftmax_grad(std::shared_ptr<op_t> t)
        {
            auto& output = get_output<la::gpu::tensor_like<double>>(t);
            auto& grad = get_grad<la::gpu::tensor_like<double>>(t);

            auto ch = get_child(t, 0);
            auto& ch_t = get_output<la::gpu::tensor_like<double>>(ch);

            if (ch->grad_needed && ch->grad == nullptr) {
                la::gpu::tensor<double> g;
                la::gpu::resize_as(g, ch_t);
                ch->grad = std::make_shared<la::gpu::tensor<double>>(std::move(g));
            }

            auto& result = get_grad<la::gpu::tensor_like<double>>(ch);

            if (ch->grad_needed) {
                op::gpu::ilogsoftmax_grad(result, grad, output);
            }
        }

        void dot_eval(std::shared_ptr<op_t> t)
        {
            auto& v = get_output<la::gpu::tensor_like<double>>(get_child(t, 0));
            auto& u = get_output<la::gpu::tensor_like<double>>(get_child(t, 1));

            t->output = std::make_shared<double>(la::gpu::dot(v, u));
        }

        void dot_grad(std::shared_ptr<op_t> t)
        {
            auto c0 = get_child(t, 0);
            auto c1 = get_child(t, 1);

            auto& v = get_output<la::gpu::tensor_like<double>>(c0);
            auto& u = get_output<la::gpu::tensor_like<double>>(c1);

            assert(v.vec_size() == u.vec_size());

            double grad = get_grad<double>(t);

            if (c0->grad_needed && c0->grad == nullptr) {
                la::gpu::tensor<double> g;
                la::gpu::resize_as(g, u);
                c0->grad = std::make_shared<la::gpu::tensor<double>>(std::move(g));
            }

            auto& v_grad = get_grad<la::gpu::tensor_like<double>>(c0);

            if (c0->grad_needed) {
                la::gpu::axpy(v_grad, grad, u);
            }

            if (c1->grad_needed && c1->grad == nullptr) {
                la::gpu::tensor<double> g;
                la::gpu::resize_as(g, v);
                c1->grad = std::make_shared<la::gpu::tensor<double>>(std::move(g));
            }

            auto& u_grad = get_grad<la::gpu::tensor_like<double>>(c1);

            if (c1->grad_needed) {
                la::gpu::axpy(u_grad, grad, v);
            }
        }

        void row_cat_eval(std::shared_ptr<op_t> t)
        {
            auto& g = *t->graph;
            assert(g.adj[t->id].size() > 0);

            unsigned int rows = g.adj[t->id].size();

            auto& v0 = get_output<la::gpu::tensor_like<double>>(get_child(t, 0));

            if (t->output == nullptr) {
                std::vector<unsigned int> sizes = v0.sizes();
                std::vector<unsigned int> new_sizes;
                new_sizes.push_back(rows);
                new_sizes.insert(new_sizes.end(), sizes.begin(), sizes.end());

                la::gpu::tensor<double> z;
                z.resize(new_sizes);
                t->output = std::make_shared<la::gpu::tensor<double>>(std::move(z));
            }

            auto& z = get_output<la::gpu::tensor_like<double>>(t);

            la::gpu::weak_matrix<double> m { z.data(), rows, v0.vec_size() };

            for (int i = 0; i < m.rows(); ++i) {
                auto& vi = get_output<la::gpu::tensor_like<double>>(get_child(t, i));

                assert(vi.vec_size() == m.cols());

                la::gpu::weak_tensor<double> mi (z.data() + i * m.cols(), {m.cols()});

                la::gpu::copy(mi, vi);
            }
        }

        void row_cat_grad(std::shared_ptr<op_t> t)
        {
            auto& g = *t->graph;

            auto& z = autodiff::get_grad<la::gpu::tensor_like<double>>(t);

            assert(z.size(0) == g.adj[t->id].size());

            la::gpu::weak_matrix<double> m { z.data(), z.size(0), z.vec_size() / z.size(0) };

            for (int i = 0; i < m.rows(); ++i) {
                auto c = get_child(t, i);

                auto& v = autodiff::get_output<la::gpu::tensor_like<double>>(c);

                assert(v.vec_size() == m.cols());

                if (c->grad_needed && c->grad == nullptr) {
                    la::gpu::tensor<double> g;
                    la::gpu::resize_as(g, v);
                    c->grad = std::make_shared<la::gpu::tensor<double>>(std::move(g));
                }

                auto& g = autodiff::get_grad<la::gpu::tensor_like<double>>(c);

                if (c->grad_needed) {
                    la::gpu::axpy(g.as_vector(), 1,
                        la::gpu::weak_vector<double>(z.data() + i * m.cols(), m.cols()));
                }
            }
        }

        void reshape_eval(std::shared_ptr<op_t> t)
        {
            std::vector<unsigned int>& sizes = *std::static_pointer_cast<std::vector<unsigned int>>(t->data);
            la::gpu::tensor_like<double>& input = get_output<la::gpu::tensor_like<double>>(get_child(t, 0));

            unsigned int d = (sizes.size() == 0 ? 0 : 1);
            for (int i = 0; i < sizes.size(); ++i) {
                d *= sizes[i];
            }
            assert(d <= input.vec_size());

            la::gpu::weak_tensor<double> result { input.data(), sizes };
            t->output = std::make_shared<la::gpu::weak_tensor<double>>(result);

            auto c = get_child(t, 0);

            if (c->grad_needed && c->grad == nullptr) {
                la::gpu::tensor<double> g;
                la::gpu::resize_as(g, input);
                c->grad = std::make_shared<la::gpu::tensor<double>>(std::move(g));
            }

            if (c->grad_needed) {
                auto& g = autodiff::get_grad<la::gpu::tensor_like<double>>(c);
                la::gpu::weak_tensor<double> wg { g.data(), sizes };
                t->grad = std::make_shared<la::gpu::weak_tensor<double>>(wg);
            }
        }

        void reshape_grad(std::shared_ptr<op_t> t)
        {
        }

        void resize_as_eval(std::shared_ptr<op_t> t)
        {
            auto c = get_child(t, 0);

            if (t->output == nullptr) {
                auto& c_t = get_output<la::gpu::tensor_like<double>>(c);

                double value = *std::static_pointer_cast<double>(t->data);

                la::gpu::tensor<double> w;
                la::gpu::resize_as(w, c_t, value);

                t->output = std::make_shared<la::gpu::tensor<double>>(std::move(w));
            }
        }

        void resize_as_grad(std::shared_ptr<op_t> t)
        {
        }

        void rep_row_to_eval(std::shared_ptr<op_t> t)
        {
            auto& u = get_output<la::gpu::tensor_like<double>>(get_child(t, 0));
            auto& v = get_output<la::gpu::tensor_like<double>>(get_child(t, 1));

            assert(v.vec_size() % u.vec_size() == 0);

            if (t->output == nullptr) {
                la::gpu::tensor<double> w;
                std::vector<unsigned int> sizes = u.sizes();
                std::vector<unsigned int> new_sizes;
                new_sizes.push_back(v.vec_size() / u.vec_size());
                new_sizes.insert(new_sizes.end(), sizes.begin(), sizes.end());
                w.resize(new_sizes);

                t->output = std::make_shared<la::gpu::tensor<double>>(std::move(w));
            }

            auto& w = get_output<la::gpu::tensor_like<double>>(t);

            la::gpu::weak_matrix<double> w_mat {w.data(), v.vec_size() / u.vec_size(), u.vec_size()};

            la::gpu::vector<double> one;
            one.resize(v.vec_size() / u.vec_size(), 1);

            la::gpu::outer_prod(w_mat, one, u.as_vector());
        }

        void rep_row_to_grad(std::shared_ptr<op_t> t)
        {
            auto u_op = get_child(t, 0);

            auto& u = get_output<la::gpu::tensor_like<double>>(u_op);
            auto& v = get_output<la::gpu::tensor_like<double>>(get_child(t, 1));

            if (u_op->grad_needed && u_op->grad == nullptr) {
                la::gpu::tensor<double> g;
                la::gpu::resize_as(g, u);
                u_op->grad = std::make_shared<la::gpu::tensor<double>>(std::move(g));
            }

            auto& g_w = get_grad<la::gpu::tensor_like<double>>(t);
            auto& g_u = get_grad<la::gpu::tensor_like<double>>(u_op);

            if (u_op->grad_needed) {
                la::gpu::weak_matrix<double> z {
                    g_w.data(), g_w.vec_size() / u.vec_size(), u.vec_size()};

                la::gpu::vector<double> one;
                one.resize({g_w.vec_size() / u.vec_size()}, 1);

                la::gpu::lmul(g_u.as_vector(), one, z);
            }
        }

        void rep_col_to_eval(std::shared_ptr<op_t> t)
        {
            auto& u = get_output<la::gpu::tensor_like<double>>(get_child(t, 0));
            auto& v = get_output<la::gpu::tensor_like<double>>(get_child(t, 1));

            assert(v.vec_size() % u.vec_size() == 0);

            if (t->output == nullptr) {
                la::gpu::tensor<double> w;
                std::vector<unsigned int> sizes = u.sizes();
                sizes.push_back(v.vec_size() / u.vec_size());
                w.resize(sizes);
                t->output = std::make_shared<la::gpu::tensor<double>>(std::move(w));
            }

            auto& w = get_output<la::gpu::tensor_like<double>>(t);

            la::gpu::vector<double> one;
            one.resize(v.vec_size() / u.vec_size(), 1);

            la::gpu::outer_prod(w.as_matrix(), u.as_vector(), one);
        }

        void rep_col_to_grad(std::shared_ptr<op_t> t)
        {
            auto& u = get_output<la::gpu::tensor_like<double>>(get_child(t, 0));
            auto& v = get_output<la::gpu::tensor_like<double>>(get_child(t, 1));

            assert(v.vec_size() % u.vec_size() == 0);

            auto u_op = get_child(t, 0);

            if (u_op->grad_needed && u_op->grad == nullptr) {
                la::gpu::tensor<double> g;
                la::gpu::resize_as(g, u);
                u_op->grad = std::make_shared<la::gpu::tensor<double>>(std::move(g));
            }

            auto& g_w = get_grad<la::gpu::tensor_like<double>>(t);
            auto& g_u = get_grad<la::gpu::tensor_like<double>>(u_op);

            if (u_op->grad_needed) {
                la::gpu::vector<double> one;
                one.resize(v.vec_size() / u.vec_size(), 1);
                
                la::gpu::mul(g_u.as_vector(), g_w.as_matrix(), one);
            }
        }

        void dropout_mask_eval(std::shared_ptr<op_t> t)
        {
            auto& u = get_output<la::gpu::tensor_like<double>>(get_child(t, 0));

            double prob;
            std::default_random_engine *gen;
            std::tie(prob, gen) = *std::static_pointer_cast<
                std::tuple<double, std::default_random_engine*>>(t->data);

            la::cpu::tensor<double> w;
            w.resize(u.sizes());

            std::bernoulli_distribution bernoulli { 1 - prob };

            double *w_data = w.data();

            for (int i = 0; i < w.vec_size(); ++i) {
                w_data[i] = bernoulli(*gen) / (1 - prob);
            }

            t->output = std::make_shared<la::gpu::tensor<double>>(la::gpu::tensor<double>(w));
        }

        void dropout_mask_grad(std::shared_ptr<op_t> t)
        {
        }

    }
}
